//
// Created by jsmith on 10/12/15.
//
//#include <hip/driver_types.h>
#include "IMT.h"

IMT::IMT(int _x, int _y, std::string fuel_model_name, std::string fuel_moisture_name)
        : Propagation(_x, _y, fuel_model_name, fuel_moisture_name){
   toa_map_ = (int*)malloc(sim_size_ * sizeof(int));
   timesteppers_ = (float*)malloc(2 * sizeof(float));
   check_ = (bool*)malloc(sim_size_ * sizeof(bool));
//   l_n_ = (float*)malloc(16 * sizeof(float));
}

IMT::~IMT(){
   // Free Host Memory
   free(toa_map_);
   free(timesteppers_);
   free(check_);
//   free(l_n_);
   // Free Device Memory
   hipFree(g_toa_map_now_);
   hipFree(g_toa_map_next_);
   hipFree(g_toa_map_step_);
   hipFree(g_timesteppers_);
   hipFree(g_l_n_);
}

bool IMT::Init(std::string fuel_file, std::string terrain_file,
               std::string canopy_height_file, std::string crown_base_height_file,
               std::string crown_bulk_density_file, float wind_x, float wind_y) {
   Propagation::Init(fuel_file, terrain_file,
                     canopy_height_file, crown_base_height_file,
                     crown_bulk_density_file, wind_x, wind_y);
   // Initialize TOA Map
   for(unsigned int i = 0; i < sim_size_; i++){
      toa_map_[i] = simulation_->ign_time_[i];
      check_[i] = false;
   }
   // Initialize TimeNow and TimeNext
   timesteppers_[1] = timesteppers_[0] = 0;
   // Populate lengths
//   for(unsigned int i = 0; i < 16; i++){
//      l_n_[i] = simulation_->l_n_[i];
//   }
   l_n_ = simulation_->l_n_;
   return true;
}

bool IMT::CopyToDevice() {
   Propagation::CopyToDevice();
   // Create Memory on Device
   hipError_t err;
   err = hipMalloc((void**) &g_toa_map_now_, sim_size_*sizeof(int));
   err = hipMalloc((void**) &g_toa_map_next_, sim_size_*sizeof(int));
   err = hipMalloc((void**) &g_toa_map_step_, sim_size_*sizeof(int));
   err = hipMalloc((void**) &g_timesteppers_, 2*sizeof(float));
   err = hipMalloc((void**) &g_l_n_, 16 * sizeof(float));
   err = hipMalloc((void**) &g_check_, sim_size_ * sizeof(bool));
   if (err != hipSuccess) {
      std::cerr << "Error Allocating Memory in IMT Class: " << hipGetErrorString(err) << std::endl;
      exit(1);
      return false;
   }

   // Copy data to device
   err = hipMemcpy(g_toa_map_now_, toa_map_, sim_size_*sizeof(int), hipMemcpyHostToDevice);
   err = hipMemcpy(g_toa_map_next_, toa_map_, sim_size_*sizeof(int), hipMemcpyHostToDevice);
   err = hipMemcpy(g_toa_map_step_, toa_map_, sim_size_*sizeof(int), hipMemcpyHostToDevice);
   err = hipMemcpy(g_timesteppers_, timesteppers_, 2 * sizeof(float), hipMemcpyHostToDevice);
   err = hipMemcpy(g_l_n_, l_n_, 16*sizeof(float), hipMemcpyHostToDevice);
   err = hipMemcpy(g_check_, check_, sim_size_*sizeof(bool), hipMemcpyHostToDevice);
   if (err != hipSuccess) {
      std::cerr << "Error Copying Memory in IMT Class: " << hipGetErrorString(err) << std::endl;
      exit(1);
      return false;
   }
}

bool IMT::RunKernel(int sim_step, int B, int T, bool crowning_flag) {
   int counter = 0;
   int terminate = -1;
//   while(terminate <= 0){
   while(counter < sim_step){
      counter++;
      // ITERATIVE MINIMAL TIME
      // Do calculations
      ItMinTime<<<B,T>>>(g_toa_map_now_,g_toa_map_next_, g_toa_map_step_, g_curspreadrate_,
                         g_timesteppers_, g_l_n_, g_check_, sim_size_,
                         sim_rows_, sim_cols_);
      // cout << "step caclulated\n";
      // Copy from output to write
//      copyKernelIMT<<<B,T>>>(g_toa_map_now_, g_toa_map_step_,
//                             g_check_, sim_size_);
      if(crowning_flag)
         TestCrownRate<<<B,T>>>(g_curspreadrate_,g_maxspreadrate_,g_intensity_modifier_,sim_size_, g_I_o_, g_RAC_);
      // Accelerate Fire
      Accelerate<<<B,T>>>(g_curspreadrate_, g_maxspreadrate_, acceleration_constant_, sim_size_ * 16, simulation_->time_step_);

      hipDeviceSynchronize();

      if(terminate < sim_size_)
         terminate = -1;

      // cout << counter <<endl;
      // Swap Pointers for loop
      int *swap = g_toa_map_now_;
      g_toa_map_now_ = g_toa_map_next_;
      g_toa_map_next_ = swap;

   }
   return true;
}

bool IMT::CopyFromDevice() {
   hipError_t err = hipMemcpy(toa_map_, g_toa_map_now_, sim_size_ * sizeof(int), hipMemcpyDeviceToHost);
   if (err != hipSuccess) {
      std::cerr << "Error copying data from GPU: " << hipGetErrorString(err) << std::endl;
      exit(1);
      return false;
   }
   return true;
}

bool IMT::WriteToFile(std::string filename) {
   std::ofstream fout;
//   std::string filename;
//   filename += simulation_->root_path_;
//   filename += "out/IMT_test.csv";
   fout.open(filename.c_str());
   for(unsigned int i = 0; i < sim_size_; i++){
      if(i % simulation_->sim_dim_x_ == 0 && i !=0){
         fout << '\n';
      }
      fout << (int) toa_map_[i] << ",";
   }
   fout.close();
   return true;
}

bool IMT::UpdateCell(int _x, int _y, int val){
//   if(_x < 0 || _y < 0 || _x > sim_rows_ || _y > sim_cols_)
//      return false;
   int cell = _x * sim_cols_ + _y;
   toa_map_[cell] = val;
   return true;
}
